#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <assert.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "common.h"

#define NUM_THREADS 256
#define bin_size (cutoff)
// We can prove, assuming the correctness checker (based on minimum distance) is correct,
// that the number of particles in each bin at any given time step is at most 6.
// Please see our report for more details.
#define MAX_PARTICLES_PER_BIN 6

extern double size;

__device__ void apply_force_gpu(particle_t &particle, particle_t &neighbor)
{
    double dx = neighbor.x - particle.x;
    double dy = neighbor.y - particle.y;
    double r2 = dx * dx + dy * dy;
    if( r2 > cutoff*cutoff )
        return;
    r2 = (r2 > min_r*min_r) ? r2 : min_r*min_r;
    double r = sqrt( r2 );

    //
    //  very simple short-range repulsive force
    //
    double coef = ( 1 - cutoff / r ) / r2 / mass;
    particle.ax += coef * dx;
    particle.ay += coef * dy;

}

__global__ void move_gpu (particle_t * particles, int n, double size)
{

    // Get thread (particle) ID
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if(tid >= n) return;

    particle_t * p = &particles[tid];
    //
    //  slightly simplified Velocity Verlet integration
    //  conserves energy better than explicit Euler method
    //
    p->vx += p->ax * dt;
    p->vy += p->ay * dt;
    p->x  += p->vx * dt;
    p->y  += p->vy * dt;

    //
    //  bounce from walls
    //
    while( p->x < 0 || p->x > size )
    {
        p->x  = p->x < 0 ? -(p->x) : 2*size-p->x;
        p->vx = -(p->vx);
    }
    while( p->y < 0 || p->y > size )
    {
        p->y  = p->y < 0 ? -(p->y) : 2*size-p->y;
        p->vy = -(p->vy);
    }

}

// Assign each particle to correct bin
__global__ void binning_gpu (particle_t *particles, int n, particle_t **bins, 
                             int *num_particles_per_bins, int nbins_side) {
   // Get thread ID
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid >= n) return;

    // Assigning bins to threads
    int bin_x_id = floor(particles[tid].x / bin_size);
    int bin_y_id = floor(particles[tid].y / bin_size);
    int bin_id = nbins_side * bin_x_id  + bin_y_id;

    // Indexing without race condition
    int thread_index_in_bin = atomicAdd(num_particles_per_bins+bin_id, 1);
    bins[MAX_PARTICLES_PER_BIN * bin_id + thread_index_in_bin] = particles + tid;
}

__global__ void compute_forces_gpu(particle_t *particles, particle_t **bins,
                                   int *num_particles_per_bins, int nbins_side, int n) {
  // Get particle ID
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid >= n) return;

  int bin_x_id = floor(particles[tid].x / bin_size);
  int bin_y_id = floor(particles[tid].y / bin_size);

  // Compute forces
  particles[tid].ax = particles[tid].ay = 0;
  for (int i = max(bin_x_id - 1, 0) ; i < min(bin_x_id + 2, nbins_side) ; i ++) {
        for (int j = max(bin_y_id - 1, 0) ; j < min(bin_y_id + 2, nbins_side) ; j ++) {
            int bid = nbins_side * i + j;
            for (int k = 0 ; k < num_particles_per_bins[bid] ; k ++) {
                apply_force_gpu(particles[tid], *(bins[bid * MAX_PARTICLES_PER_BIN + k]));
            }
        }
    }
}

int main(int argc, char **argv) {    
    // This takes a few seconds to initialize the runtime
    hipDeviceSynchronize(); 

    if (find_option(argc, argv, "-h") >= 0) {
        printf("Options:\n");
        printf("-h to see this help\n");
        printf("-n <int> to set the number of particles\n");
        printf("-o <filename> to specify the output file name\n");
        return 0;
    }
    
    int n = read_int(argc, argv, "-n", 1000);

    char *savename = read_string(argc, argv, "-o", NULL);
    
    FILE *fsave = savename ? fopen(savename, "w") : NULL;
    particle_t *particles = (particle_t*) malloc(n * sizeof(particle_t));

    // GPU particle data structure
    particle_t * d_particles;
    hipMalloc((void **) &d_particles, n * sizeof(particle_t));
    set_size(n);
    init_particles(n, particles);

    int nbins_side = ceil(size / bin_size);
    int numbins = nbins_side * nbins_side;
    particle_t **d_bins; 
    hipMalloc((void ***) &d_bins, numbins * MAX_PARTICLES_PER_BIN * sizeof(particle_t *));

    int *num_particles_per_bins;
    hipMalloc((void **) &num_particles_per_bins, numbins * sizeof(int *));

    hipDeviceSynchronize();
    double copy_time = read_timer( );

    // Copy the particles to the GPU
    hipMemcpy(d_particles, particles, n * sizeof(particle_t), hipMemcpyHostToDevice);

    hipDeviceSynchronize();
    copy_time = read_timer() - copy_time;
    
    //
    //  simulate a number of time steps
    //
    hipDeviceSynchronize();
    double simulation_time = read_timer();

    for( int step = 0; step < NSTEPS; step++ ) {
        //
        //  compute forces
        //
	    int blks = (n + NUM_THREADS - 1) / NUM_THREADS;

        hipMemset(num_particles_per_bins, 0, numbins * sizeof(int));

		// assign particles to bins
        binning_gpu<<<blks, NUM_THREADS>>>(d_particles, n, d_bins, num_particles_per_bins, nbins_side);

        // compute the forces
        compute_forces_gpu<<<blks, NUM_THREADS>>>(d_particles, d_bins, num_particles_per_bins, nbins_side, n);
        
        // move particles
        move_gpu<<<blks, NUM_THREADS>>>(d_particles, n, size);
        
        //
        //  save if necessary
        //
        if(fsave && (step%SAVEFREQ) == 0) {
            // Copy the particles back to the CPU
            hipMemcpy(particles, d_particles, n * sizeof(particle_t), hipMemcpyDeviceToHost);
            save(fsave, n, particles);
        }
    }
    hipDeviceSynchronize();
    simulation_time = read_timer() - simulation_time;
    
    printf("CPU-GPU copy time = %g seconds\n", copy_time);
    printf("n = %d, simulation time = %g seconds\n", n, simulation_time );
    
    
    free(particles);
    hipFree(d_particles);
    hipFree(d_bins);
    hipFree(num_particles_per_bins);
    if(fsave) fclose(fsave);
    
    return 0;
}
